
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
//#include <cuda.h>
#include <stdio.h>

using namespace std;

///////////////////////////////////////////////////////////////////////////////
	
void print(std::vector<float> &vec) 
{
  for (size_t i = 0; i < vec.size(); ++i) {
     cerr << vec[i] << " ";
  }
  cerr << endl;
}

///////////////////////////////////////////////////////////////////////////////
__global__
void kernelHelloWorld()
{
  printf("Hello world\n");
}
///////////////////////////////////////////////////////////////////////////////
__global__
void kernelSquare(float *vec)
{
  for (size_t i = 0; i < 10; ++i) {
    vec[i] = vec[i] * vec[i];
  }
}
///////////////////////////////////////////////////////////////////////////////
__global__
void kernelParallelSquare(float *vec, int size)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size) {
    printf("gridDim.x=%i  blockDim.x=%i blockIdx.x=%i threadIdx.x=%i i=%i \n", gridDim.x, blockDim.x, blockIdx.x, threadIdx.x, i);
    vec[i] = vec[i] * vec[i];
  }
}

///////////////////////////////////////////////////////////////////////////////
__global__
void kernelReduce(float *vec, int size)
{
  for (int i = 1; i < size; ++i) {
    vec[0] += vec[i];
  }
}

///////////////////////////////////////////////////////////////////////////////
__global__
void kernelReduceAtomic(float *vec, int size)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size) {
    atomicAdd(&vec[0], vec[i]);
  }  
}

///////////////////////////////////////////////////////////////////////////////
__global__
void kernelReduceParallel(float *vec, int size, int half)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < half) {
    vec[i] += vec[i+half];
    if ((i == half - 1) && (i + half + 2 == size)) {
       vec[i] += vec[i+half+1];
    }
  }
}

///////////////////////////////////////////////////////////////////////////////
__global__
void kernelReduceParallel2(float *vec, int size)
{
  int half = size / 2;
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  while (half > 0) {
    if (i < half) {
      vec[i] += vec[i+half];
      if ((i == half - 1) && (i + half + 2 == size)) {
        vec[i] += vec[i+half+1];
      }
    }

    size = half;
    half = size / 2;
  }
}

///////////////////////////////////////////////////////////////////////////////

int main()
{
  cerr << "Starting" << endl;

  int NUM = 10;
  vector<float> h_vec(NUM);
  for (size_t i = 0; i < NUM; ++i) {
     h_vec[i] = i;
  }
  print(h_vec); cerr << endl;

  float *d_array;
  hipMalloc(&d_array, sizeof(float) * NUM);
  hipMemcpy(d_array, h_vec.data(), sizeof(float) * NUM, hipMemcpyHostToDevice);

  //kernel1<<<1, 1000>>>();
  //kernel2<<<1, 1>>>(d_array);
  //kernel3<<<3, 4>>>(d_array, NUM);
  //kernel4<<<1, 1>>>(d_array, NUM);
  //kernel5<<<1, 10>>>(d_array, NUM);

  //kernelReduceParallel<<<1, 5>>>(d_array, 10, 5);
  //kernelReduceParallel<<<1, 2>>>(d_array, 5, 2);
  //kernelReduceParallel<<<1, 1>>>(d_array, 2, 1);
  /*
  int size = 10;
  int half = size / 2;
  while (half > 0) {
    kernelReduceParallel<<<1, half>>>(d_array, size, half);

    size = half;
    half = size / 2;
  }
  */
  kernelReduceParallel2<<<1, 5>>>(d_array, 10);

  if ( hipSuccess != hipGetLastError()) {
    cerr << "kernel didn't run" << endl;
    abort();
  }
  
  int ret = hipDeviceSynchronize();
  if (ret) {
    cerr << "kernel ran but produced an error" << endl;
    abort();
  }

  hipMemcpy(h_vec.data(), d_array, sizeof(float) * NUM, hipMemcpyDeviceToHost);

  print(h_vec); cerr << endl;

  cerr << "Finished" << endl;
}
